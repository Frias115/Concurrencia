#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : practica2Cuda.cu
 Author      : Ana Casado, Roberto Frias & Sergio Rosello
 Version     : 1.0
 Copyright   : MIT
 Description : Optimizaciones usando GPU
 ============================================================================
 */

#include "practica2Cuda.h"
//#define LADO 5000 //lado de la matriz
//2249 max extraño

using namespace std;

//lamada de la gpu para multiplicar los vectores
__device__ float kernel_multiplicarVectores(int lado, float* fila, float* columna){
	if(threadIdx.x == 0 && threadIdx.y == 0){
		//printf("Multiplicando vectores\n");
	}
	float resultadoAux = 0;
	for(int i = 0; i<lado; i++){
		//printf("fila: %.0f columna: %.0f", fila[i], columna[i]);
		resultadoAux += fila[i] * columna[i];
	}

	//printf("resultado: %.0f \n", resultadoAux);


	return resultadoAux;

}

//llamada a la gpu para multiplicar matrices
__global__ void kernel_multiplicarMatrices(int lado, float** matriz1, float** matriz2, float** resultado){
	//printf("estoy multiplicando\n");
	int fila = blockIdx.x * blockDim.y + threadIdx.y;
	int columna = blockIdx.y * blockDim.x + threadIdx.x;


 	//HABRIA QUE HACERLO CON MEMORIA COMPARTIDA PARA QUE LOS ACCESOS SEAN MUCHO MENOS https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
 	//LA IDEA ES PILLAR UNAS CUANTAS FILAS O COLUMNAS Y UTILIZARLAS PARA REALIZAR TODOS LOS CALCULOS, DE MANERA QUE PILLAMOS UN BLOQUE DE X COLUMNAS QUE QUEPA 
 	// EN LOS 40K QUE ACEPTAN 1024 THREADS(32*32) Y ASI UTILIZARLA EN TODOS LOS CALCULOS

 	//TAMBIEN HAY QUE COLOCAR LA PARTE DE CREAR MATRICES EN OTRO LADO

 	//NO ESTARIA MAL TAMPOCO INTENTAR ARREGLARLO PARA QUE FUNCIONE CON NUESTRAS MATRICES ANTERIORES, AUNQUE TAMPOCO ES NECESARIO

 	//HAY QUE VOLVER A HACER TODOS LOS CALCULOS EN LAS OTRAS ENTREGAS PARA DAR BIEN LOS TIEMPOS Y EXPLICAR PORQUE DAN QUE ERA POR LO DEL NUMERO DE CORES DEL
 	//PROCESADOR 

	//control de errores del thread
	if((fila >= lado) || (columna >= lado)){
		//printf("ha ocurrido un error en multiplicacion\n");
		return;
		}

	resultado[fila][columna] = kernel_multiplicarVectores(lado, matriz1[fila], matriz2[columna]);
	//printf("cuda thread %d %d %.0f \n",fila,columna,resultado[fila][columna]);
}

paqueteTrabajo* mainCuda(paqueteTrabajo* paquete){


	DEBUG_TIME_INIT;
	DEBUG_TIME_START;

	//Inicialización de la semilla para los números aleatorios.
	srand(time(NULL));

	//bool leerTraspuesta = true;

	//CARGA E INICIALIZACION DE LAS MATRICES
	//CPU
	float** matriz1_host = (float**)paquete->datosUno;
	float** matriz2_host = (float**)paquete->datosDos;
	float** matrizResultado_host;

	//NEXO (memoria intermedia)
	float** matriz1_nexo;
	float** matriz2_nexo;
	float** matrizResultado_nexo;

	//GPU
	float** matriz1_device;
	float** matriz2_device;
	float** matrizResultado_device;

	int size = paquete->numeroRealColumnasACalcular;

	//leemos de fichero binario
	//leerDatosBin(argv[2], &matriz1_host, leerTraspuesta, size);
	//leerDatosBin("nuevadosmilidentidad.bin", &matriz2_host, leerTraspuesta);

	//IMPRIME LAS MATRICES GENERADAS
	//printf("Se van a generar matrices de %d X %d : \n", LADO, LADO);
	//printf("MATRIZ A: \n\n");
	//imprimirMatriz(matriz1_host);
	//printf("MATRIZ B: \n\n");
	//imprimirMatriz(matriz2_host);

	//se hace la traspuesta de la segunda matriz para poder multiplicarla
	//leerDatosBin(argv[3], &matriz2_host, !leerTraspuesta, size);
	//printf("MATRIZ B traspuesta: \n\n");
	//imprimirMatriz(matriz2_host);

	//Reserva para el resultado del host
	matrizResultado_host = (float**)malloc(size * sizeof(float*));
	for(int i=0; i < size; i++){
		matrizResultado_host[i] = (float*)malloc(size * sizeof(float));
	}

	//Reserva de la memoria intermedia
	matriz1_nexo = (float**)malloc(size * sizeof(float*));
	matriz2_nexo = (float**)malloc(size * sizeof(float*));
	matrizResultado_nexo = (float**)malloc(size * sizeof(float*));

	//Reserva de memoria en GPU
	hipError_t err1 = hipMalloc((void**)&matriz1_device, sizeof(float*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));

	err1 = hipMalloc((void**)&matriz2_device, sizeof(float*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));
	err1 = hipMalloc((void**)&matrizResultado_device, sizeof(float*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));

	//Reserva de memoria para cada uno de los arrays intermedios
	for(int i = 0; i < size; i++){
		err1 = hipMalloc((void**)&matriz1_nexo[i], sizeof(float)* size);
		//printf("matriz1_nexo Run Kernel: %s \n", hipGetErrorString(err1));
		err1 = hipMalloc((void**)&matriz2_nexo[i], sizeof(float)* size);
		//printf("matriz2_nexo Run Kernel: %s \n", hipGetErrorString(err1));
		hipMalloc((void**)&(matrizResultado_nexo[i]), sizeof(float)* size);
	}

	//Copia el contenido de los arrays de CPU a los arrays de la matriz intermedia
	for(int i = 0; i < size; i++){
			err1 = hipMemcpy(matriz1_nexo[i], matriz1_host[i], size * sizeof(float),hipMemcpyHostToDevice);
			//printf("cudaMemcoy matriz2_host1 a nexo1 Run Kernel: %s \n", hipGetErrorString(err1));
			err1 = hipMemcpy(matriz2_nexo[i], matriz2_host[i], size * sizeof(float),hipMemcpyHostToDevice);
			//printf("cudaMemcoy matriz2_host2 a nexo2 Run Kernel: %s \n", hipGetErrorString(err1));
	}

	//copia el contenido del array de punteros de CPU a GPU
	err1 = hipMemcpy(matriz1_device, matriz1_nexo, size * sizeof(float*),hipMemcpyHostToDevice);
	//printf("copia de cpu a gpu array de punteros matriz1 Run Kernel: %s \n", hipGetErrorString(err1));
	err1 = hipMemcpy(matriz2_device, matriz2_nexo, size * sizeof(float*),hipMemcpyHostToDevice);
	//printf("copia de cpu a gpu array de punteros matriz2 Run Kernel: %s \n", hipGetErrorString(err1));
	hipMemcpy(matrizResultado_device, matrizResultado_nexo, size * sizeof(float*),hipMemcpyHostToDevice);

	//Operaciones en GPU:
	// tamBloque = 32 porque los kernels proporcionan las instrucciones en warps (32 threads)
	//entonces tiene que ser multiplo de 32 para no despediciar threads.
	// dimensionGrid -> (LADO / tamBloque) + 1 para calcular el numero de bloques para la x y para y 1 para la z
	// dimensionBlock -> numero de threads por cada bloque (32*32 = 1024 threads)
	// https://codeyarns.com/2011/02/16/cuda-dim3/
	// http://www.icl.utk.edu/~mgates3/docs/cuda.html
	int tamBloque = 32;

	dim3 dimensionGrid = dim3((int)(size / tamBloque) + 1, (int)(size / tamBloque) + 1, 1);
	dim3 dimensionBlock = dim3(tamBloque, tamBloque, 1);

	//printf("Antes de multiplicar\n");
	//hace la multiplicacion en GPU
	{
		DEBUG_TIME_INIT;
		DEBUG_TIME_START;

		kernel_multiplicarMatrices <<<dimensionGrid,dimensionBlock>>>(size, matriz1_device, matriz2_device, matrizResultado_device);

		//Para que espere hasta que todos los threads terminen (CUDA THREADS SYNCRONIZE)
		hipError_t error = hipDeviceSynchronize();
		printf("Thread synchronization: %s \n", hipGetErrorString(error));

		//CUDA MEMCHECK SE ASEGURA DE LOS ACCESOS A MEMORIA ESTEN BIEN
		DEBUG_TIME_END;
		DEBUG_PRINT_FINALTIME("Tiempo multiplicarMatrices(): \n\t");

	}

	//pasamos el resultado de device al host
	for(int i = 0; i < size; i++){
		err1 = hipMemcpy(matrizResultado_host[i], matrizResultado_nexo[i], size * sizeof(float),hipMemcpyDeviceToHost);
		//printf("copia de gpu a cpu final Run Kernel: %s \n", hipGetErrorString(err1));
	}

	hipDeviceSynchronize();

	//imprime la matriz resultado na vez copiada al host
	//printf("El resultado es: \n");
	//guardarMatriz(matrizResultado_host, size);

	paquete->resultado = (int**)matrizResultado_host;

	//LIBERACION DE MEMORIA DE CPU Y INTERMEDIA
	for(int i = 0; i < size; i++){
		//CPU
		free(matriz1_host[i]);
		free(matriz2_host[i]);
		free(matrizResultado_host[i]);

		//Intermedia
		hipFree(matriz1_nexo[i]);
		hipFree(matriz2_nexo[i]);
		hipFree(matrizResultado_nexo[i]);
	}

	//liberacion del array de punteros
	free(matriz1_host);
	free(matriz2_host);
	free(matrizResultado_host);

	free(matriz1_nexo);
	free(matriz2_nexo);
	free(matrizResultado_nexo);

	//free GPU
	hipFree(matriz1_device);
	hipFree(matriz2_device);
	hipFree(matrizResultado_device);

	DEBUG_TIME_END;

	DEBUG_PRINT_FINALTIME("Tiempo Total: \n\t");




	return paquete;
	
}
#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : practica2Cuda.cu
 Author      : Ana Casado, Roberto Frias & Sergio Rosello
 Version     : 1.0
 Copyright   : MIT
 Description : Optimizaciones usando GPU
 ============================================================================
 */

#include "practica2Cuda.h"
//#define LADO 5000 //lado de la matriz
//2249 max extraño

using namespace std;

//lamada de la gpu para multiplicar los vectores
__device__ int kernel_multiplicarVectores(int lado, int* fila, int* columna){
	if(threadIdx.x == 0 && threadIdx.y == 0){
		//printf("Multiplicando vectores\n");
	}
	int resultadoAux = 0;
	for(int i = 0; i<lado; i++){
		//printf("fila: %.0f columna: %.0f", fila[i], columna[i]);
		resultadoAux += fila[i] * columna[i];
	}

	//printf("resultado: %.0f \n", resultadoAux);

	return resultadoAux;

}

//llamada a la gpu para multiplicar matrices
__global__ void kernel_multiplicarMatrices(int lado, int** matriz1, int** matriz2, int** resultado){
	//printf("estoy multiplicando\n");

	int fila = blockIdx.x * blockDim.y + threadIdx.y;
	int columna = blockIdx.y * blockDim.x + threadIdx.x;

	//control de errores del thread
	if((fila >= lado) || (columna >= lado)){
		//printf("ha ocurrido un error en multiplicacion\n");
		return;
		}

	resultado[fila][columna] = kernel_multiplicarVectores(lado, matriz1[fila], matriz2[columna]);
	//printf("cuda thread %d %d %.0f \n",fila,columna,resultado[fila][columna]);

}

paqueteTrabajo* mainCuda(paqueteTrabajo* paquete){

	//Inicialización de la semilla para los números aleatorios.
	srand(time(NULL));

	//bool leerTraspuesta = true;

	//CARGA E INICIALIZACION DE LAS MATRICES
	//CPU
	cout << "Recibo las matrices en CUDA" << endl;
	int** matriz1_host;
	int** matriz2_host;
	int** matrizResultado_host;

	//NEXO (memoria intermedia)
	int** matriz1_nexo;
	int** matriz2_nexo;
	int** matrizResultado_nexo;

	//GPU
	int** matriz1_device;
	int** matriz2_device;
	int** matrizResultado_device;

	int size = paquete->numeroRealColumnasACalcular;

	//leemos de fichero binario
	//leerDatosBin(argv[2], &matriz1_host, leerTraspuesta, size);
	//leerDatosBin("nuevadosmilidentidad.bin", &matriz2_host, leerTraspuesta);

	//IMPRIME LAS MATRICES GENERADAS
	//printf("Se van a generar matrices de %d X %d : \n", LADO, LADO);
	//printf("MATRIZ A: \n\n");
	//imprimirMatriz(matriz1_host);
	//printf("MATRIZ B: \n\n");
	//imprimirMatriz(matriz2_host);

	//se hace la traspuesta de la segunda matriz para poder multiplicarla
	//leerDatosBin(argv[3], &matriz2_host, !leerTraspuesta, size);
	//printf("MATRIZ B traspuesta: \n\n");
	//imprimirMatriz(matriz2_host);

	matriz1_host = (int **)malloc(sizeof(int *) * size);
	for (int i = 0; i < size; i++) {
		matriz1_host[i] = (int *)malloc(sizeof(int) * size);
	}

	matriz1_host = paquete->datosUno;

	matriz2_host = (int **)malloc(sizeof(int *) * size);
	for (int i = 0; i < size; i++) {
		matriz2_host[i] = (int *)malloc(sizeof(int) * size);
	}

	matriz2_host = paquete->datosDos;
	
	//Reserva para el resultado del host
	matrizResultado_host = (int**)malloc(size * sizeof(int*));
	for(int i=0; i < size; i++){
		matrizResultado_host[i] = (int*)malloc(size * sizeof(int));
	}

	//Reserva de la memoria intermedia
	matriz1_nexo = (int**)malloc(size * sizeof(int*));
	matriz2_nexo = (int**)malloc(size * sizeof(int*));
	matrizResultado_nexo = (int**)malloc(size * sizeof(int*));

	//Reserva de memoria en GPU
	hipError_t err1 = hipMalloc((void**)&matriz1_device, sizeof(int*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));

	err1 = hipMalloc((void**)&matriz2_device, sizeof(int*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));
	err1 = hipMalloc((void**)&matrizResultado_device, sizeof(int*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));


	//Reserva de memoria para cada uno de los arrays intermedios
	for(int i = 0; i < size; i++){
		err1 = hipMalloc((void**)&matriz1_nexo[i], sizeof(int)* size);
		//printf("matriz1_nexo Run Kernel: %s \n", hipGetErrorString(err1));
		err1 = hipMalloc((void**)&matriz2_nexo[i], sizeof(int)* size);
		//printf("matriz2_nexo Run Kernel: %s \n", hipGetErrorString(err1));
		hipMalloc((void**)&(matrizResultado_nexo[i]), sizeof(int)* size);
	}

	//Copia el contenido de los arrays de CPU a los arrays de la matriz intermedia
	for(int i = 0; i < size; i++){
		err1 = hipMemcpy(matriz1_nexo[i], matriz1_host[i], size * sizeof(int),hipMemcpyHostToDevice);
		//printf("cudaMemcoy matriz2_host1 a nexo1 Run Kernel: %s \n", hipGetErrorString(err1));
		err1 = hipMemcpy(matriz2_nexo[i], matriz2_host[i], size * sizeof(int),hipMemcpyHostToDevice);
		//printf("cudaMemcoy matriz2_host2 a nexo2 Run Kernel: %s \n", hipGetErrorString(err1));
	}

	//copia el contenido del array de punteros de CPU a GPU
	err1 = hipMemcpy(matriz1_device, matriz1_nexo, size * sizeof(int*),hipMemcpyHostToDevice);
	//printf("copia de cpu a gpu array de punteros matriz1 Run Kernel: %s \n", hipGetErrorString(err1));
	err1 = hipMemcpy(matriz2_device, matriz2_nexo, size * sizeof(int*),hipMemcpyHostToDevice);
	//printf("copia de cpu a gpu array de punteros matriz2 Run Kernel: %s \n", hipGetErrorString(err1));
	hipMemcpy(matrizResultado_device, matrizResultado_nexo, size * sizeof(int*),hipMemcpyHostToDevice);

	//Operaciones en GPU:
	// tamBloque = 32 porque los kernels proporcionan las instrucciones en warps (32 threads)
	//entonces tiene que ser multiplo de 32 para no despediciar threads.
	// dimensionGrid -> (LADO / tamBloque) + 1 para calcular el numero de bloques para la x y para y 1 para la z
	// dimensionBlock -> numero de threads por cada bloque (32*32 = 1024 threads)
	// https://codeyarns.com/2011/02/16/cuda-dim3/
	// http://www.icl.utk.edu/~mgates3/docs/cuda.html
	int tamBloque = 32;

	dim3 dimensionGrid = dim3((int)(size / tamBloque) + 1, (int)(size / tamBloque) + 1, 1);
	dim3 dimensionBlock = dim3(tamBloque, tamBloque, 1);

	//printf("Antes de multiplicar\n");
	//hace la multiplicacion en GPU
	kernel_multiplicarMatrices <<<dimensionGrid,dimensionBlock>>>(size, matriz1_device, matriz2_device, matrizResultado_device);

	//Para que espere hasta que todos los threads terminen (CUDA THREADS SYNCRONIZE)
	hipError_t error = hipDeviceSynchronize();
	printf("Thread synchronization: %s \n", hipGetErrorString(error));

	//pasamos el resultado de device al host
	for(int i = 0; i < size; i++){
		err1 = hipMemcpy(matrizResultado_host[i], matrizResultado_nexo[i], size * sizeof(int),hipMemcpyDeviceToHost);
		//printf("copia de gpu a cpu final Run Kernel: %s \n", hipGetErrorString(err1));
	}

	hipDeviceSynchronize();

	//imprime la matriz resultado na vez copiada al host
	//printf("El resultado es: \n");
	//guardarMatriz(matrizResultado_host, size);

	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			paquete->resultado[i][j] = matrizResultado_host[i][j];
		}
	}

	//LIBERACION DE MEMORIA DE CPU Y INTERMEDIA
	for(int i = 0; i < size; i++){
		//CPU
		free(matriz1_host[i]);
		free(matriz2_host[i]);
		free(matrizResultado_host[i]);

		//Intermedia
		hipFree(matriz1_nexo[i]);
		hipFree(matriz2_nexo[i]);
		hipFree(matrizResultado_nexo[i]);
	}

	//liberacion del array de punteros
	free(matriz1_host);
	free(matriz2_host);
	free(matrizResultado_host);

	free(matriz1_nexo);
	free(matriz2_nexo);
	free(matrizResultado_nexo);

	//free GPU
	hipFree(matriz1_device);
	hipFree(matriz2_device);
	hipFree(matrizResultado_device);

	return paquete;
	
}
#include "hip/hip_runtime.h"
#include "debug_time.h"
#include "/usr/include/openmpi/mpi.h"
#include <stdio.h>
#include <stdlib.h>
#include <complex>

using namespace std;

int NUM_THREADS = 1;

struct paqueteTrabajo {
	int filaInicial;
	int numeroRealFilasACalcular;
	int numeroRealColumnasACalcular;
    int **datosUno;
    int **datosDos;
    int **resultado;
};

paqueteTrabajo* mainCuda(paqueteTrabajo* paquete);
__host__ __device__ float kernel_multiplicarVectores(int lado, float* fila, float* columna);
__global__ void kernel_multiplicarMatrices(int lado, float** matriz1, float** matriz2, float** resultado);

class Matriz{

public:
	int numFilas;
	int numColumnas;
	int **datos;

	Matriz(int numFilas, int numColumnas);

	Matriz(string nombre, bool leerTraspuesta);

	void guardarMatriz(string nombre);

	void imprimirMatriz();

	void reservarMemoria();

	Matriz *multiplicarMatrices(Matriz *segundaMatriz, int numeroThreads, int numeroEsclavos);

	void enviarDatosAEsclavo(int parteMatriz, Matriz *segundaMatriz, int numeroEsclavos);

	~Matriz();
};

Matriz::Matriz(string nombre, bool leerTraspuesta){

	FILE *fichero = fopen(nombre.c_str(), "r+");
	if(fichero == NULL) {
		printf("Error");
		return;
	}

	fread(&this->numFilas, sizeof(char), sizeof(int), fichero);
	fread(&this->numColumnas, sizeof(char), sizeof(int), fichero);

	this->reservarMemoria();

	if(leerTraspuesta){
		for (int i = 0; i < this->numColumnas; i++){
			for (int j = 0; j < this->numFilas; j++){
				fread(&(this->datos[j][i]), sizeof(char), sizeof(int), fichero);
			}
		}
	}
	else{
		for (int i = 0; i < this->numFilas; i++) 
			fread(this->datos[i], sizeof(float), this->numColumnas, fichero);
	}

	fclose(fichero);
}

Matriz::Matriz(int numFilas, int numColumnas){
	this->numFilas = numFilas;
	this->numColumnas = numColumnas;

	this->datos = (int **)malloc(sizeof(int*)*numFilas);
	for(int i = 0; i < numFilas; i++){
		this->datos[i] = (int*)malloc(sizeof(int)*numColumnas);
	}
}

void Matriz::reservarMemoria(){
	this->datos = (int **)malloc(sizeof(int*)*this->numFilas);
	for(int i = 0; i < this->numFilas; i++){
		this->datos[i] = (int*)malloc(sizeof(int)*this->numColumnas);
	}
}

void Matriz::guardarMatriz(string nombre){

	FILE* fich = fopen(nombre.c_str(), "w");
	if(fich == NULL) {
		printf("Error");
		return;
	}
	
	fwrite(&this->numFilas, sizeof(char), sizeof(int), fich);
	fwrite(&this->numColumnas, sizeof(char), sizeof(int), fich);
	for(int i = 0; i < this->numFilas; i++){
		for(int j = 0; j < this->numColumnas; j++){
			fwrite(&this->datos[i][j], sizeof(char), sizeof(int), fich);
		}
	}
	fclose(fich);
}

Matriz *Matriz::multiplicarMatrices(Matriz *segundaMatriz, int numeroThreads, int numeroEsclavos) {
	Matriz *resultado = new Matriz(this->numFilas, this->numColumnas);
	cout << "Tengo "  << numeroEsclavos-1 << " esclavos"<< endl;
	if( numeroThreads <= 0){
		NUM_THREADS = 1;
	} else
	{
		NUM_THREADS = numeroThreads;
	}

	for (int i = 1; i < numeroEsclavos; i++){
		enviarDatosAEsclavo((i -1), segundaMatriz, (numeroEsclavos -1));
		
		printf("\n Mensaje enviado a %d \n", i);

	}

	cout << "Los esclavos ya tienen la informacion " << endl;

	for (int i = 1; i < numeroEsclavos; i++){
		MPI_Status status;
		
		int filaInicial;
		int numeroRealFilasACalcular;
		int numeroRealColumnasACalcular;

		//recibo
		MPI_Recv(&filaInicial, sizeof(int), MPI_BYTE, i, 0, MPI_COMM_WORLD, &status);
		MPI_Recv(&numeroRealFilasACalcular, sizeof(int), MPI_BYTE, i, 0, MPI_COMM_WORLD, &status);
		MPI_Recv(&numeroRealColumnasACalcular, sizeof(int), MPI_BYTE, i, 0, MPI_COMM_WORLD, &status);

		int** aux = (int **)malloc(sizeof(int*)*numeroRealColumnasACalcular);
		for(int j = 0; j < numeroRealColumnasACalcular; j++){
			aux[j] = (int*)malloc(sizeof(int)*numeroRealColumnasACalcular);
		}

		for (int j = 0; j < numeroRealColumnasACalcular; j++){
			MPI_Recv(aux[j], numeroRealColumnasACalcular, MPI_INT, i, 0, MPI_COMM_WORLD, &status);
		}

		cout << "Recibo el paquete del esclavo " << i << endl;
		//junto resultados
		for (int j = 0; j < numeroRealColumnasACalcular; j++){
			resultado->datos[j+filaInicial] = aux[j+filaInicial];
		}
		free(aux);
	}

	return resultado;
}

void Matriz::enviarDatosAEsclavo(int parteMatriz, Matriz *segundaMatriz, int numeroEsclavos){

	int filaInicial;
	int numeroRealFilasACalcular;
	int numeroRealColumnasACalcular;

	float aux = (float)this->numFilas / (float)numeroEsclavos;

	if(ceil(aux) <= (this->numFilas - (ceil(aux) * parteMatriz))){
		numeroRealFilasACalcular = ceil(aux);
		filaInicial = numeroRealFilasACalcular*parteMatriz;
	}else{
		numeroRealFilasACalcular = (this->numFilas - (ceil(aux) * parteMatriz));
		filaInicial = (ceil(aux))*parteMatriz;
	}
	numeroRealColumnasACalcular = this->numColumnas;
	
	//Envio informacion a los esclavos
	MPI_Send(&NUM_THREADS, sizeof(int), MPI_BYTE, parteMatriz+1, 0, MPI_COMM_WORLD);
	MPI_Send(&filaInicial, sizeof(int), MPI_BYTE, parteMatriz+1, 0, MPI_COMM_WORLD);
	MPI_Send(&numeroRealFilasACalcular, sizeof(int), MPI_BYTE, parteMatriz+1, 0, MPI_COMM_WORLD);
	MPI_Send(&numeroRealColumnasACalcular, sizeof(int), MPI_BYTE, parteMatriz+1, 0, MPI_COMM_WORLD);

	for (int i = 0; i < numeroRealFilasACalcular; ++i){
		MPI_Send(this->datos[i+filaInicial], numeroRealColumnasACalcular, MPI_INT, parteMatriz+1, 0, MPI_COMM_WORLD);
	}

	for (int i = 0; i < this->numFilas; ++i){
		MPI_Send(segundaMatriz->datos[i], numeroRealColumnasACalcular, MPI_INT, parteMatriz+1, 0, MPI_COMM_WORLD);
	}
}

void Matriz::imprimirMatriz(){
	for(int i = 0; i < this->numFilas; i++){
		for(int j = 0; j < this->numColumnas; j++){
			cout << this->datos[i][j] << " ";
		}
		cout << endl;
	}
}

Matriz::~Matriz(){
	for (int i = 0; i < this->numColumnas; ++i){
	free(this->datos[i]);
	}
	free(this->datos);
}

//Añadir parametros necesarios para leer (m origen y nombre de matriz resultado)
void maestro(int rank, int nproc, Matriz *m1, Matriz *m2, int numeroThreads){

	int source = rank;

	DEBUG_TIME_INIT;
	DEBUG_TIME_START;

	{
		DEBUG_TIME_INIT;
		DEBUG_TIME_START;
		//dividir matriz // crear paquetes de trabajo
			//por cada esclavo
			//enviar submatriz
			//por cada esclavo
		//recibir resultados

		//escribir resultado
		cout << "Ejecuto multiplicarMatrices" << endl;
		Matriz *resultado = m1->multiplicarMatrices(m2, numeroThreads, nproc);
		cout << "Acaba multiplicarMatrices" << endl;
		DEBUG_TIME_END;
		DEBUG_PRINT_FINALTIME("Tiempo multiplicarMatrices(): \n\t");
		resultado->guardarMatriz("resultado");
	}

	DEBUG_TIME_END;

	DEBUG_PRINT_FINALTIME("Tiempo Total: \n\t");
}

void esclavo(int rank, int nproc){

	int dest = rank;
	MPI_Status status;

	paqueteTrabajo *paquete = (paqueteTrabajo *)malloc(sizeof(paqueteTrabajo));
	int numeroThreads = 0;
	int filaInicial = 0;
	int numeroRealFilasACalcular = 0;
	int numeroRealColumnasACalcular = 0;

	//Recibe la informacion 
	cout << rank << " -> Espera a los datos" << endl;
	MPI_Recv(&numeroThreads, sizeof(int), MPI_BYTE, 0, 0, MPI_COMM_WORLD, &status);
	MPI_Recv(&filaInicial, sizeof(int), MPI_BYTE, 0, 0, MPI_COMM_WORLD, &status);
	MPI_Recv(&numeroRealFilasACalcular, sizeof(int), MPI_BYTE, 0, 0, MPI_COMM_WORLD, &status);
	MPI_Recv(&numeroRealColumnasACalcular, sizeof(int), MPI_BYTE, 0, 0, MPI_COMM_WORLD, &status);

	Matriz *matriz1 = new Matriz(numeroRealFilasACalcular, numeroRealColumnasACalcular);
	Matriz *matriz2 = new Matriz(numeroRealColumnasACalcular, numeroRealColumnasACalcular); 
	Matriz *resultado = new Matriz(numeroRealColumnasACalcular, numeroRealColumnasACalcular);

	for (int i = 0; i < numeroRealFilasACalcular; ++i){
		MPI_Recv(matriz1->datos[i], numeroRealColumnasACalcular, MPI_INT, 0, 0, MPI_COMM_WORLD, &status);
	}

	for (int i = 0; i < numeroRealColumnasACalcular; ++i){
		MPI_Recv(matriz2->datos[i], numeroRealColumnasACalcular, MPI_INT, 0, 0, MPI_COMM_WORLD, &status);
	}

	cout << rank << " -> Recibe los datos" << endl;

	//Mete en un paquete de trabajo la informacion
	paquete->filaInicial = filaInicial;
	paquete->numeroRealFilasACalcular = numeroRealFilasACalcular;
	paquete->numeroRealColumnasACalcular = numeroRealColumnasACalcular;
	paquete->datosUno = matriz1->datos;
	paquete->datosDos = matriz2->datos;
	paquete->resultado = resultado->datos;

	paqueteTrabajo *paqueteAux;
	//Multiplicar (estatica/dinamica/gpu)
	cout << rank << " -> Multiplica los datos" << endl;
	paqueteAux = mainCuda(paquete);
	cout << rank << " -> Recibe el resultado de multiplicar" << endl;

	//Enviar informacion de vuelta al maestro
	cout << "paqueteAux->filaInicial " << paqueteAux->filaInicial << endl;
	cout << "paqueteAux->numeroRealFilasACalcular" << paqueteAux->numeroRealFilasACalcular << endl;
	cout << rank << " -> Devuelve datos al maestro" << endl;
	MPI_Send(&paqueteAux->filaInicial, sizeof(int), MPI_BYTE, 0, 0, MPI_COMM_WORLD);
	MPI_Send(&paqueteAux->numeroRealFilasACalcular, sizeof(int), MPI_BYTE, 0, 0, MPI_COMM_WORLD);
	MPI_Send(&paqueteAux->numeroRealColumnasACalcular, sizeof(int), MPI_BYTE, 0, 0, MPI_COMM_WORLD);


	for (int i = 0; i < numeroRealColumnasACalcular; i++){
		MPI_Send(paqueteAux->resultado[i], numeroRealColumnasACalcular, MPI_INT, 0, 0, MPI_COMM_WORLD);
	}

	free(paqueteAux);
	cout << rank << " -> Acaba" << endl;

}

int main(int arg, char **argv){

	//Para c
	//Para compilar: mpicc ejemplo

	//Para c++
	//para compilar: mpiCC ejemplo.cpp -o ejemplo

	//Para ejecutar: mpirun -np 4 (ruta del ejecutable) (parametros)

	//Para ejecutar en cluster: mpirun -np 4 (ruta del ejecutable) -hostfile hostfile.txt

	//Orden parametros: <MATRIZ1> <MATRIZ2> <NUM_THREADS>

	int nproc, rank;
	MPI_Init(&arg, &argv);
	//MPI_COMM_WORLD: Comunicador
	MPI_Comm_size(MPI_COMM_WORLD, &nproc);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	MPI_Status status;
	cout << "Leo las matrices" << endl;
	bool leerTraspuesta = true;

	//Hay que enviar la matriz por cachos

	//leer matrices
	Matriz *m1 = new Matriz(argv[1], !leerTraspuesta);
	Matriz *m2 = new Matriz(argv[2], leerTraspuesta);
	int numeroThreads = atoi(argv[3]);

	cout << "Termino de leer las matrices" << endl;

	if(rank == 0){
		cout << rank << " -> Entra al maestro" << endl;
		maestro(rank, nproc, m1, m2, numeroThreads);
	}
	else //if(rank == 1)
	{
		cout << rank << " -> Entra al esclavo" << endl;
		esclavo(rank, nproc);
	}
	MPI_Finalize();
	return 0;
}

/*

si se quejade specified application es que esta mal la ruta


mpirun -np 4 -hostfile hostfile.txt /(ruta del programa)

hostfile.txt:

10.4.0.29
10.4.0.44

*/


/*
 ============================================================================
 Name        : practica2Cuda.cu
 Author      : Ana Casado, Roberto Frias & Sergio Rosello
 Version     : 1.0
 Copyright   : MIT
 Description : Optimizaciones usando GPU
 ============================================================================
 */


//lamada de la gpu para multiplicar los vectores
__device__ float kernel_multiplicarVectores(int lado, float* fila, float* columna){
	if(threadIdx.x == 0 && threadIdx.y == 0){
		//printf("Multiplicando vectores\n");
	}
	float resultadoAux = 0;
	for(int i = 0; i<lado; i++){
		//printf("fila: %.0f columna: %.0f", fila[i], columna[i]);
		resultadoAux += fila[i] * columna[i];
	}

	//printf("resultado: %.0f \n", resultadoAux);


	return resultadoAux;

}

//llamada a la gpu para multiplicar matrices
__global__ void kernel_multiplicarMatrices(int lado, float** matriz1, float** matriz2, float** resultado){
	//printf("estoy multiplicando\n");
	int fila = blockIdx.x * blockDim.y + threadIdx.y;
	int columna = blockIdx.y * blockDim.x + threadIdx.x;


 	//HABRIA QUE HACERLO CON MEMORIA COMPARTIDA PARA QUE LOS ACCESOS SEAN MUCHO MENOS https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
 	//LA IDEA ES PILLAR UNAS CUANTAS FILAS O COLUMNAS Y UTILIZARLAS PARA REALIZAR TODOS LOS CALCULOS, DE MANERA QUE PILLAMOS UN BLOQUE DE X COLUMNAS QUE QUEPA 
 	// EN LOS 40K QUE ACEPTAN 1024 THREADS(32*32) Y ASI UTILIZARLA EN TODOS LOS CALCULOS

 	//TAMBIEN HAY QUE COLOCAR LA PARTE DE CREAR MATRICES EN OTRO LADO

 	//NO ESTARIA MAL TAMPOCO INTENTAR ARREGLARLO PARA QUE FUNCIONE CON NUESTRAS MATRICES ANTERIORES, AUNQUE TAMPOCO ES NECESARIO

 	//HAY QUE VOLVER A HACER TODOS LOS CALCULOS EN LAS OTRAS ENTREGAS PARA DAR BIEN LOS TIEMPOS Y EXPLICAR PORQUE DAN QUE ERA POR LO DEL NUMERO DE CORES DEL
 	//PROCESADOR 

	//control de errores del thread
	if((fila >= lado) || (columna >= lado)){
		//printf("ha ocurrido un error en multiplicacion\n");
		return;
		}

	resultado[fila][columna] = kernel_multiplicarVectores(lado, matriz1[fila], matriz2[columna]);
	//printf("cuda thread %d %d %.0f \n",fila,columna,resultado[fila][columna]);
}

paqueteTrabajo* mainCuda(paqueteTrabajo* paquete){


	DEBUG_TIME_INIT;
	DEBUG_TIME_START;

	//Inicialización de la semilla para los números aleatorios.
	srand(time(NULL));

	//bool leerTraspuesta = true;

	//CARGA E INICIALIZACION DE LAS MATRICES
	//CPU
	float** matriz1_host = (float**)paquete->datosUno;
	float** matriz2_host = (float**)paquete->datosDos;
	float** matrizResultado_host;

	//NEXO (memoria intermedia)
	float** matriz1_nexo;
	float** matriz2_nexo;
	float** matrizResultado_nexo;

	//GPU
	float** matriz1_device;
	float** matriz2_device;
	float** matrizResultado_device;

	int size = paquete->numeroRealColumnasACalcular;

	//leemos de fichero binario
	//leerDatosBin(argv[2], &matriz1_host, leerTraspuesta, size);
	//leerDatosBin("nuevadosmilidentidad.bin", &matriz2_host, leerTraspuesta);

	//IMPRIME LAS MATRICES GENERADAS
	//printf("Se van a generar matrices de %d X %d : \n", LADO, LADO);
	//printf("MATRIZ A: \n\n");
	//imprimirMatriz(matriz1_host);
	//printf("MATRIZ B: \n\n");
	//imprimirMatriz(matriz2_host);

	//se hace la traspuesta de la segunda matriz para poder multiplicarla
	//leerDatosBin(argv[3], &matriz2_host, !leerTraspuesta, size);
	//printf("MATRIZ B traspuesta: \n\n");
	//imprimirMatriz(matriz2_host);

	//Reserva para el resultado del host
	matrizResultado_host = (float**)malloc(size * sizeof(float*));
	for(int i=0; i < size; i++){
		matrizResultado_host[i] = (float*)malloc(size * sizeof(float));
	}

	//Reserva de la memoria intermedia
	matriz1_nexo = (float**)malloc(size * sizeof(float*));
	matriz2_nexo = (float**)malloc(size * sizeof(float*));
	matrizResultado_nexo = (float**)malloc(size * sizeof(float*));

	//Reserva de memoria en GPU
	hipError_t err1 = hipMalloc((void**)&matriz1_device, sizeof(float*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));

	err1 = hipMalloc((void**)&matriz2_device, sizeof(float*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));
	err1 = hipMalloc((void**)&matrizResultado_device, sizeof(float*)* size);
	//printf("Run Kernel: %s \n", hipGetErrorString(err1));

	//Reserva de memoria para cada uno de los arrays intermedios
	for(int i = 0; i < size; i++){
		err1 = hipMalloc((void**)&matriz1_nexo[i], sizeof(float)* size);
		//printf("matriz1_nexo Run Kernel: %s \n", hipGetErrorString(err1));
		err1 = hipMalloc((void**)&matriz2_nexo[i], sizeof(float)* size);
		//printf("matriz2_nexo Run Kernel: %s \n", hipGetErrorString(err1));
		hipMalloc((void**)&(matrizResultado_nexo[i]), sizeof(float)* size);
	}

	//Copia el contenido de los arrays de CPU a los arrays de la matriz intermedia
	for(int i = 0; i < size; i++){
			err1 = hipMemcpy(matriz1_nexo[i], matriz1_host[i], size * sizeof(float),hipMemcpyHostToDevice);
			//printf("cudaMemcoy matriz2_host1 a nexo1 Run Kernel: %s \n", hipGetErrorString(err1));
			err1 = hipMemcpy(matriz2_nexo[i], matriz2_host[i], size * sizeof(float),hipMemcpyHostToDevice);
			//printf("cudaMemcoy matriz2_host2 a nexo2 Run Kernel: %s \n", hipGetErrorString(err1));
	}

	//copia el contenido del array de punteros de CPU a GPU
	err1 = hipMemcpy(matriz1_device, matriz1_nexo, size * sizeof(float*),hipMemcpyHostToDevice);
	//printf("copia de cpu a gpu array de punteros matriz1 Run Kernel: %s \n", hipGetErrorString(err1));
	err1 = hipMemcpy(matriz2_device, matriz2_nexo, size * sizeof(float*),hipMemcpyHostToDevice);
	//printf("copia de cpu a gpu array de punteros matriz2 Run Kernel: %s \n", hipGetErrorString(err1));
	hipMemcpy(matrizResultado_device, matrizResultado_nexo, size * sizeof(float*),hipMemcpyHostToDevice);

	//Operaciones en GPU:
	// tamBloque = 32 porque los kernels proporcionan las instrucciones en warps (32 threads)
	//entonces tiene que ser multiplo de 32 para no despediciar threads.
	// dimensionGrid -> (LADO / tamBloque) + 1 para calcular el numero de bloques para la x y para y 1 para la z
	// dimensionBlock -> numero de threads por cada bloque (32*32 = 1024 threads)
	// https://codeyarns.com/2011/02/16/cuda-dim3/
	// http://www.icl.utk.edu/~mgates3/docs/cuda.html
	int tamBloque = 32;

	dim3 dimensionGrid = dim3((int)(size / tamBloque) + 1, (int)(size / tamBloque) + 1, 1);
	dim3 dimensionBlock = dim3(tamBloque, tamBloque, 1);

	//printf("Antes de multiplicar\n");
	//hace la multiplicacion en GPU
	{
		DEBUG_TIME_INIT;
		DEBUG_TIME_START;

		kernel_multiplicarMatrices <<<dimensionGrid,dimensionBlock>>>(size, matriz1_device, matriz2_device, matrizResultado_device);

		//Para que espere hasta que todos los threads terminen (CUDA THREADS SYNCRONIZE)
		hipError_t error = hipDeviceSynchronize();
		printf("Thread synchronization: %s \n", hipGetErrorString(error));

		//CUDA MEMCHECK SE ASEGURA DE LOS ACCESOS A MEMORIA ESTEN BIEN
		DEBUG_TIME_END;
		DEBUG_PRINT_FINALTIME("Tiempo multiplicarMatrices(): \n\t");

	}

	//pasamos el resultado de device al host
	for(int i = 0; i < size; i++){
		err1 = hipMemcpy(matrizResultado_host[i], matrizResultado_nexo[i], size * sizeof(float),hipMemcpyDeviceToHost);
		//printf("copia de gpu a cpu final Run Kernel: %s \n", hipGetErrorString(err1));
	}

	hipDeviceSynchronize();

	//imprime la matriz resultado na vez copiada al host
	//printf("El resultado es: \n");
	//guardarMatriz(matrizResultado_host, size);

	paquete->resultado = (int**)matrizResultado_host;

	//LIBERACION DE MEMORIA DE CPU Y INTERMEDIA
	for(int i = 0; i < size; i++){
		//CPU
		free(matriz1_host[i]);
		free(matriz2_host[i]);
		free(matrizResultado_host[i]);

		//Intermedia
		hipFree(matriz1_nexo[i]);
		hipFree(matriz2_nexo[i]);
		hipFree(matrizResultado_nexo[i]);
	}

	//liberacion del array de punteros
	free(matriz1_host);
	free(matriz2_host);
	free(matrizResultado_host);

	free(matriz1_nexo);
	free(matriz2_nexo);
	free(matrizResultado_nexo);

	//free GPU
	hipFree(matriz1_device);
	hipFree(matriz2_device);
	hipFree(matrizResultado_device);

	DEBUG_TIME_END;

	DEBUG_PRINT_FINALTIME("Tiempo Total: \n\t");




	return paquete;
	
}

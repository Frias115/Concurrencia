/*
 ============================================================================
 Name        : practica2Cuda.cu
 Author      : Sergio Rosello & César Gil
 Version     : 0.1
 Copyright   : If you copy you will fail
 Description : Optimizaciones usando GPU
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#define LADO 2000 //lado de la matriz

using namespace std;

void inicializarMatrizConsecutiva(float ** matriz, int tamFilas, int tamColumnas){
	int i, j;
	for (i = 0; i < tamFilas; i++)
		for (j = 0; j < tamColumnas; j++)
			matriz[i][j] = i*tamFilas + j;
}

void inicializarMatrizAleatoria(float **matriz, int tamFilas, int tamColumnas) {
	int i, j;
	for (i = 0; i < tamFilas; i++)
		for (j = 0; j < tamColumnas; j++)
			matriz[i][j] = rand() % 10;
	//matriz[i][j] = 1;
}

void crearMatriz(char* nombre)
{

	int i = 0;
	//int j = 0;

	//Inicializamos memoria para la matriz.
	float **mat = (float **)malloc(sizeof(float *) * LADO);
	for (i = 0; i < LADO; i++) {
		mat[i] = (float *)malloc(sizeof(float) * LADO);
	}

	inicializarMatrizAleatoria(mat, LADO, LADO);

	//Abrimos el fichero binario mat en modo de escritura.
	FILE* fich_bin = fopen(nombre, "w");

	//Escribimos en el archivo binario apuntado por fich_bin

	//Volcamos los datos de la matriz en el archivo binario apuntado por fich_bin
	for (i = 0; i < LADO; i++)
		fwrite(mat[i], sizeof(int), LADO, fich_bin);
	fclose(fich_bin);

	//Liberamos memoria de cada uno de los elementos de la matriz.
	for (i = 0; i < LADO; i++) {
		free(mat[i]);
	}

	//Liberamos memoria de la matriz.
	free(mat);

}

//Imprime la matriz recibida
void imprimirMatriz(float **matriz) {

	FILE* fich = fopen("resultado.bin", "w");
	if(fich == NULL) {
		//printf("Error");
	} else{
		//int aux = LADO;
		//fwrite(&(aux), sizeof(char), sizeof(int), fich);
		//fwrite(&(aux), sizeof(char), sizeof(int), fich);
		int aux = 1;
		for (int i = 0; i < LADO; i++) {
			for (int j = 0; j < LADO; j++){
				//printf("%.0f ", matriz[i][j]);
				fwrite(&(matriz[j][i]), sizeof(float),1, fich);
			}
			//printf("\n");
		}
	}

	fclose(fich);
}

//lee la matriz de un fichero binario
void leerDatosBin(char *nombreFichero, float ***datos, bool leerTraspuesta) {
	FILE* fichero = fopen(nombreFichero, "r");
	//Funciones accesibles: fclose, fread, fwrite
	float **datosLeidos;
	int numFilas, numColumnas;
	int i = 0, j = 0;


	//Inicializamos un array para guardar todos los datos que leemos del fichero.
	datosLeidos = (float **)malloc(LADO * sizeof(float*));

	//multiplicamos por 4 (bytes que ocupa un float)
	for (int i = 0; i < LADO; i++)
		datosLeidos[i] = (float*)malloc(sizeof(float)*LADO);

	if(!leerTraspuesta)
		for (int i = 0; i < LADO; i++)
			for (int j = 0; j < LADO; j++)
				fread(&(datosLeidos[i][j]), sizeof(float), 1, fichero);

	else //Leer la matriz de una forma traspuesta.
		for (int i = 0; i < LADO; i++)
			for (int j = 0; j < LADO; j++)
				fread(&(datosLeidos[j][i]), sizeof(float), 1, fichero);


	//Derreferenciación.
	(*datos) = datosLeidos;
	fclose(fichero);
}

//lamada de la gpu para multiplicar los vectores
__device__ float multiplicarVectores(int lado, float* fila, float* columna){
	if(threadIdx.x == 0 && threadIdx.y == 0)
		printf("Multiplicando vectores\n");
	float resultadoAux = 0;
	for(int i = 0; i<lado; i++){
		resultadoAux += fila[i] * columna[i];
	}

	return resultadoAux;

}

//llamada a la gpu para multiplicar matrices
__global__ void kernel_multiplicarMatrices(int lado, float** matriz1, float** matriz2, float** resultado){
	//printf("estoy multiplicando\n");
	int fila = blockIdx.x * blockDim.y + threadIdx.y;
	int columna = blockIdx.y * blockDim.x + threadIdx.x;

	//control de errores del thread
	if((fila >= lado) || (columna >= lado)){
		//printf("ha ocurrido un error en multiplicacion\n");
		return;
		}

	resultado[fila][columna] = multiplicarVectores(lado, matriz1[fila], matriz2[columna]);
	//printf("cuda thread %d %d %.0f \n",fila,columna,resultado[fila][columna]);
}



int main(){

	//Inicialización de la semilla para los números aleatorios.
	srand(time(NULL));

	bool leerTraspuesta = true;

	//CARGA E INICIALIZACION DE LAS MATRICES
	//CPU
	float** matriz1_host;
	float** matriz2_host;
	float** matrizResultado_host;

	//NEXO (memoria intermedia)
	float** matriz1_nexo;
	float** matriz2_nexo;
	float** matrizResultado_nexo;

	//GPU
	float** matriz1_device;
	float** matriz2_device;
	float** matrizResultado_device;

	//leemos de fichero binario
	leerDatosBin("nuevadosmilaleatoria.bin", &matriz1_host, leerTraspuesta);
	leerDatosBin("nuevadosmilidentidad.bin", &matriz2_host, leerTraspuesta);

	//IMPRIME LAS MATRICES GENERADAS
	//printf("Se van a generar matrices de %d X %d : \n", LADO, LADO);
	//printf("MATRIZ A: \n\n");
	//imprimirMatriz(matriz1_host);
	//printf("MATRIZ B: \n\n");
	//imprimirMatriz(matriz2_host);

	//se hace la traspuesta de la segunda matriz para poder multiplicarla
	leerDatosBin("nuevadosmilidentidad.bin", &matriz2_host, !leerTraspuesta);
	//printf("MATRIZ B traspuesta: \n\n");
	//imprimirMatriz(matriz2_host);

	//Reserva para el resultado del host
	matrizResultado_host = (float**)malloc(LADO * sizeof(float*));
	for(int i=0; i < LADO; i++){
		matrizResultado_host[i] = (float*)malloc(LADO * sizeof(float));
	}

	//Reserva de la memoria intermedia
	matriz1_nexo = (float**)malloc(LADO * sizeof(float*));
	matriz2_nexo = (float**)malloc(LADO * sizeof(float*));
	matrizResultado_nexo = (float**)malloc(LADO * sizeof(float*));

	//Reserva de memoria en GPU
	hipError_t err1 = hipMalloc((void**)&matriz1_device, sizeof(float*)* LADO);
	//printf("Run Kernel: %s \n", cudaGetErrorString(err1));

	err1 = hipMalloc((void**)&matriz2_device, sizeof(float*)* LADO);
	//printf("Run Kernel: %s \n", cudaGetErrorString(err1));
	err1 = hipMalloc((void**)&matrizResultado_device, sizeof(float*)* LADO);
	//printf("Run Kernel: %s \n", cudaGetErrorString(err1));

	//Reserva de memoria para cada uno de los arrays intermedios
	for(int i = 0; i < LADO; i++){
		err1 = hipMalloc((void**)&matriz1_nexo[i], sizeof(float)* LADO);
		//printf("matriz1_nexo Run Kernel: %s \n", cudaGetErrorString(err1));
		err1 = hipMalloc((void**)&matriz2_nexo[i], sizeof(float)* LADO);
		//printf("matriz2_nexo Run Kernel: %s \n", cudaGetErrorString(err1));
		hipMalloc((void**)&(matrizResultado_nexo[i]), sizeof(float)* LADO);
	}

	//Copia el contenido de los arrays de CPU a los arrays de la matriz intermedia
	for(int i = 0; i < LADO; i++){
			err1 = hipMemcpy(matriz1_nexo[i], matriz1_host[i], LADO * sizeof(float),hipMemcpyHostToDevice);
			//printf("cudaMemcoy matriz2_host1 a nexo1 Run Kernel: %s \n", cudaGetErrorString(err1));
			err1 = hipMemcpy(matriz2_nexo[i], matriz2_host[i], LADO * sizeof(float),hipMemcpyHostToDevice);
			//printf("cudaMemcoy matriz2_host2 a nexo2 Run Kernel: %s \n", cudaGetErrorString(err1));
	}

	//copia el contenido del array de punteros de CPU a GPU
	err1 = hipMemcpy(matriz1_device, matriz1_nexo, LADO * sizeof(float*),hipMemcpyHostToDevice);
	//printf("copia de cpu a gpu array de punteros matriz1 Run Kernel: %s \n", cudaGetErrorString(err1));
	err1 = hipMemcpy(matriz2_device, matriz2_nexo, LADO * sizeof(float*),hipMemcpyHostToDevice);
	//printf("copia de cpu a gpu array de punteros matriz2 Run Kernel: %s \n", cudaGetErrorString(err1));
	hipMemcpy(matrizResultado_device, matrizResultado_nexo, LADO * sizeof(float*),hipMemcpyHostToDevice);

	//Operaciones en GPU:
	// tamBloque = 32 porque los kernels proporcionan las instrucciones en warps (32 threads)
	//entonces tiene que ser multiplo de 32 para no despediciar threads.
	// dimensionGrid -> (LADO / tamBloque) + 1 para calcular el numero de bloques para la x y para y 1 para la z
	// dimensionBlock -> numero de threads por cada bloque (32*32 = 1024 threads)
	// https://codeyarns.com/2011/02/16/cuda-dim3/
	// http://www.icl.utk.edu/~mgates3/docs/cuda.html
	int tamBloque = 32;

	dim3 dimensionGrid = dim3((int)(LADO / tamBloque) + 1, (int)(LADO / tamBloque) + 1, 1);
	dim3 dimensionBlock = dim3(tamBloque, tamBloque, 1);

	//printf("Antes de multiplicar\n");
	//hace la multiplicacion en GPU
	kernel_multiplicarMatrices <<<dimensionGrid,dimensionBlock>>>(LADO, matriz1_device, matriz2_device, matrizResultado_device);

	//Para que espere hasta que todos los threads terminen (CUDA THREADS SYNCRONIZE)
	hipError_t error = hipDeviceSynchronize();
	//printf("Thread synchronization: %s \n", cudaGetErrorString(error));

	//pasamos el resultado de device al host
	for(int i = 0; i < LADO; i++){
		err1 = hipMemcpy(matrizResultado_host[i], matrizResultado_nexo[i], LADO * sizeof(float),hipMemcpyDeviceToHost);
		//printf("copia de gpu a cpu final Run Kernel: %s \n", cudaGetErrorString(err1));
	}

	//imprime la matriz resultado na vez copiada al host
	//printf("El resultado es: \n");
	imprimirMatriz(matrizResultado_host);

	//LIBERACION DE MEMORIA DE CPU Y INTERMEDIA
	for(int i = 0; i < LADO; i++){
		//CPU
		free(matriz1_host[i]);
		free(matriz2_host[i]);
		free(matrizResultado_host[i]);

		//Intermedia
		hipFree(matriz1_nexo[i]);
		hipFree(matriz2_nexo[i]);
		hipFree(matrizResultado_nexo[i]);
	}

	//liberacion del array de punteros
	free(matriz1_host);
	free(matriz2_host);
	free(matrizResultado_host);

	free(matriz1_nexo);
	free(matriz2_nexo);
	free(matrizResultado_nexo);

	//free GPU
	hipFree(matriz1_device);
	hipFree(matriz2_device);
	hipFree(matrizResultado_device);

}

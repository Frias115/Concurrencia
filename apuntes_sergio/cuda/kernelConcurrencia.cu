#include "hip/hip_runtime.h"
/* Aquí ya tenemos las matrices en memoria de la GPU.
Lo del doble puntero en los parámetros es opcional. Podemos hacerlo como queramos.
*/

__global__ void multMatrix(float** m1, float** m2, float** result, int numFilas, int numColumnas){
    // Solo vamos a utilizar el eje X de los threads de CUDA
    int globalId;
    globalId = blockId.x * blockDim.x + threadIdx.x; // Creo que falta algo que quiere que pongamos nosotros
    
    if(globalId > (numFilas * numColumnas)){
        return;
    }

    for(int i = 0; i < numFilas; i++){
        result[i][j] = 0;
        for(int j = 0; j < numColumnas; j++0){
            for(int k = 0; k < numFilas; k++){
                result[i][j] += m1[i][k] * m2[j][k];    // La segunda matriz debe estar transpuesta
            }
        }
    }
}


int main(int argc, char **argv){
    int numFilas = 4;
    int numColumnas = 4;
    // Leemos las matrices
    float m1[4][4] = {{0,1,2,3}, {0,1,2,3}, {0,1,2,3}, {0,1,2,3}};
    float m2[4][4] = {{0,1,2,3}, {0,1,2,3}, {0,1,2,3}, {0,1,2,3}};
    float mres[4][4] = {{0,0,0,0}, {0,0,0,0}, {0,0,0,0}, {0,0,0,0}};


    // Nos llevamos las matrices desde la CPU a la GPU
    float** d_m1;
    float** d_m2;
    float** d_mres;

    // HostDevice (matrices intermedias)
    float** hd_m1;
    float** hd_m2;
    float** hd_mres;

    hd_m1 = (float**)malloc(sizeof(float*) * numFilas);
    hd_m2 = (float**)malloc(sizeof(float*) * numFilas);
    hd_mres = (float**)malloc(sizeof(float*) * numFilas);

    // Reservamos memoria para las matrices intermedias en la GPU
    for(int i = 0; i < numFilas; i++){
       hipMalloc(&(hd_m1[i]), sizeof(float*) * numFilas));
       hipMalloc(&(hd_m2[i]), sizeof(float*) * numFilas));
       hipMalloc(&(hd_mres[i]), sizeof(float*) * numFilas));
    }

    // Reserva en la GPU
    hipMalloc(&(d_m1), sizeof(float*) * numFilas);
    hipMalloc(&(d_m2), sizeof(float*) * numFilas);
    hipMalloc(&(d_mres), sizeof(float*) * numFilas);

    // Copiamos a la GPU
    hipMemcpy(d_m1, hd_m1, sizeof(float*) * numFilas, cudaDeviceHostToDevice);
    hipMemcpy(d_m2, hd_m2, sizeof(float*) * numFilas, cudaDeviceHostToDevice);
    hipMemcpy(d_mres, hd_mres, sizeof(float*) * numFilas, cudaDeviceHostToDevice);

    // Copiar a la CPU los resultados. No hace falta la matriz resultado porque ahora mismo está vacía.
    for(int i = 0; i < numFilas; i++){
        hipMemcpy(hd_m1[i], d_m1[i], sizeof(float*) * numFilas, hipMemcpyHostToDevice);
        hipMemcpy(hd_m2[i], d_m2[i], sizeof(float*) * numFilas, hipMemcpyHostToDevice);
    }

    // Número de bloques y threads y lanzar la aplicación. TO-DO.
    int numBlock, numThreads;   // Podríamos usar un máximo de 1024 threads. Con menos threads habrá menos accesos a RAM (y por tanto, mejor).
    // Por ejemplo
    numThreads = 1;
    numBlock = 1;
    multMatrix<<<numBlock, numThreads>>>(d_m1, d_m2, d_mres, numFilas, numColumnas);

    for(int i = 0; i < numFilas; i++){
        hipMemcpy(mres[i], d_mres[i], sizeof(float*) * numFilas, hipMemcpyDeviceToHost);
    }

    // Sincronizamos
    hipDeviceSynchronize();

    for(int i = 0; i < numFilas; i++){
        printf("\n");
        for(int j = 0; j < numColumnas; j++){
            printf("%f ", mres[i][j]);
        }
    }

    getc(stdin);

    // Faltan los cudafree. TO-DO.
}

Como control de errores, comprobar que las matrices van a caber en la GPU.